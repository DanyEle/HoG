#include "hip/hip_runtime.h"
#include <stdio.h>

#include "file_operations.c"

#define STRING_BUFFER_SIZE 1024




#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
      {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}


//Input: - rgb image contained in the 'rgb' array
//		 - buffer size: the size of the RGB image
//Output: gray, an array containing the gray-scale image
int rgbToGray(byte *rgbImage, byte **grayImage, int gray_size)
{
    // Take size for gray image and allocate memory. Just one dimension for gray-scale image
    *grayImage = (byte*) malloc(sizeof(byte) * gray_size);

    // Make pointers for iteration
    byte *p_rgb = rgbImage;
    byte *p_gray = *grayImage;

    // Calculate the value for every pixel in gray
    for(int i=0; i<gray_size; i++)
    {
    	//Formula according to: https://stackoverflow.com/questions/17615963/standard-rgb-to-grayscale-conversion
        *p_gray = 0.30*p_rgb[0] + 0.59*p_rgb[1] + 0.11*p_rgb[2];
        p_rgb += 3;
        p_gray++;
    }

    return gray_size;
}




// CUDA //kernel to convert an image to gray-scale
//gray-image's memory needs to be pre-allocated
__global__ void rgb_img_to_gray( byte * dev_r_vec, byte * dev_g_vec, byte * dev_b_vec, byte * dev_gray_image, int gray_size)
{
    //Get the id of thread within a block
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	while(tid < gray_size)
	{
		//r, g, b pixels
		byte p_r = dev_r_vec[tid];
		byte p_g = dev_g_vec[tid];
		byte p_b = dev_b_vec[tid];

		//Formula according to: https://stackoverflow.com/questions/17615963/standard-rgb-to-grayscale-conversion
		dev_gray_image[tid] = 0.30 * p_r + 0.59*p_g + 0.11*p_b;
    	tid += blockDim.x * gridDim.x;
	}
}








int main (void)
{
	//###########1. STEP - LOAD THE IMAGE, ITS HEIGHT, WIDTH AND CONVERT IT TO RGB FORMAT#########

		//Specify the input image. Formats supported: png, jpg, GIF.
		const char * fileInputName = "imgs_in/lena.png";

		const char * spaceDiv = " ";
		const char * fileOutputRGB = "imgs_out/image.rgb";
		const char *pngStrings[4] = {"convert ", fileInputName, spaceDiv, fileOutputRGB};
		const char * strPngToRGB = arrayStringsToString(pngStrings, 4, STRING_BUFFER_SIZE);

		printf("Loading input image [%s] \n", fileInputName);

		//actually execute the conversion from PNG to RGB, as that format is required for the program
		int status_conversion = system(strPngToRGB);

		if(status_conversion != 0)
		{
			printf("Conversion of input PNG image to RGB was not successful. Program aborting.");
			return -1;
		}
		printf("Converted input image to RGB [%s] \n", fileOutputRGB);

		//get the height and width of the input image
		int width = 0;
		int height = 0;

		getImageSize(fileInputName, &width, &height);

		printf("Size of the loaded image : width=%d height=%d \n", width, height);

		//Three dimensions because the input image is in colored format(R,G,B)
		int rgb_size = width * height * 3;
		printf("Total amount of pixels in RGB input image is [%d] \n", rgb_size);
		//Used as a buffer for all pixels of the image
		byte * rgb_image;

		//Load up the input image in RGB format into one single flattened array (rgbImage)
		readFile(fileOutputRGB, &rgb_image, rgb_size);

		//########2. step - convert RGB image to gray-scale

	    int gray_size = rgb_size / 3;

	    byte * rVector, * gVector, * bVector;

	    //now take the RGB image vector and create three separate arrays for the R,G,B dimensions
	    getDimensionFromRGBVec(0, rgb_image,  &rVector, gray_size);
	    getDimensionFromRGBVec(1, rgb_image,  &gVector, gray_size);
	    getDimensionFromRGBVec(2, rgb_image,  &bVector, gray_size);

	    //allocate memory on the device for the r,g,b vectors
	    byte * dev_r_vec, * dev_g_vec, * dev_b_vec;
	    HANDLE_ERROR ( hipMalloc((void **)&dev_r_vec , gray_size*sizeof(byte)));
	    HANDLE_ERROR ( hipMalloc((void **)&dev_g_vec, gray_size*sizeof(byte)));
	    HANDLE_ERROR ( hipMalloc((void **)&dev_b_vec, gray_size*sizeof(byte)));

	    //copy the content of the r,g,b vectors from the host to the device
	    HANDLE_ERROR (hipMemcpy (dev_r_vec , rVector , gray_size*sizeof(byte), hipMemcpyHostToDevice));
	    HANDLE_ERROR (hipMemcpy (dev_g_vec , gVector , gray_size*sizeof(byte), hipMemcpyHostToDevice));
	    HANDLE_ERROR (hipMemcpy (dev_b_vec , bVector , gray_size*sizeof(byte), hipMemcpyHostToDevice));

	    //allocate memory on the device for the output gray image
	    byte * dev_gray_image;
	    HANDLE_ERROR ( hipMalloc((void **)&dev_gray_image, gray_size*sizeof(byte)));

	    //actually run the kernel
	    rgb_img_to_gray <<< 512, 512>>> (dev_r_vec, dev_g_vec, dev_b_vec, dev_gray_image, gray_size) ;
	    //__global__ void rgb_img_to_gray( byte * dev_r_vec, byte * dev_g_vec, byte * dev_b_vec, byte * dev_gray_image, int gray_size)


	    //run the rgb_to_gray kernel
	    //rgb_to_gray <<< height * 3, width*3>>> (dev_rgb_image , &dev_gray_image , gray_size) ;

}
