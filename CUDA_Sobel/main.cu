#include "hip/hip_runtime.h"
#include <stdio.h>
#include <chrono>


#include "functions.c"

#define STRING_BUFFER_SIZE 1024

#define SOBEL_OP_SIZE 9
#include "string.h"
#include "stdlib.h"
#include "math.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>



#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
      {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE);
    }
}


__global__ void contour(byte *dev_sobel_h, byte *dev_sobel_v, int gray_size, byte *dev_contour_img)
{
	int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
	int tid_y = threadIdx.y + blockIdx.y * blockDim.y;

	int tid = abs(tid_x - tid_y);


    // Performed on every pixel in parallel to calculate the contour image
    while(tid < gray_size)
    {
        dev_contour_img[tid] = (byte) sqrt(pow((double)dev_sobel_h[tid], 2.0) + pow((double)dev_sobel_v[tid], 2.0));

    	tid += blockDim.x * gridDim.x + blockDim.y * gridDim.y;

    }
}

//called from 'it_conv' function
__device__ int convolution(byte *X, int *Y, int c_size)
{
    int sum = 0;

    for(int i=0; i < c_size; i++) {
        sum += X[i] * Y[c_size-i-1];
    }

    return sum;
}

//called from 'it_conv' function
__device__ void makeOpMem(byte *buffer, int buffer_size, int width, int cindex, byte *op_mem)
{
    int bottom = cindex-width < 0;
    int top = cindex+width >= buffer_size;
    int left = cindex % width == 0;
    int right = (cindex+1) % width == 0;

    op_mem[0] = !bottom && !left  ? buffer[cindex-width-1] : 0;
    op_mem[1] = !bottom           ? buffer[cindex-width]   : 0;
    op_mem[2] = !bottom && !right ? buffer[cindex-width+1] : 0;

    op_mem[3] = !left             ? buffer[cindex-1]       : 0;
    op_mem[4] = buffer[cindex];
    op_mem[5] = !right            ? buffer[cindex+1]       : 0;

    op_mem[6] = !top && !left     ? buffer[cindex+width-1] : 0;
    op_mem[7] = !top              ? buffer[cindex+width]   : 0;
    op_mem[8] = !top && !right    ? buffer[cindex+width+1] : 0;
}






__global__ void it_conv(byte * buffer, int buffer_size, int width, int * dev_op, byte *dev_res)
{
    // Temporary memory for each pixel operation
    byte op_mem[SOBEL_OP_SIZE];
    memset(op_mem, 0, SOBEL_OP_SIZE);
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
	int tid_y = threadIdx.y + blockIdx.y * blockDim.y;

	//simple linearization
	int tid = abs(tid_x - tid_y);

    // Make convolution for every pixel. Each pixel --> one thread.
    while(tid < buffer_size)
    {
        // Make op_mem
        makeOpMem(buffer, buffer_size, width, tid, op_mem);

        dev_res[tid] = (byte) abs(convolution(op_mem, dev_op, SOBEL_OP_SIZE));
        /*
         * The abs function is used in here to avoid storing negative numbers
         * in a byte data type array. It wouldn't make a different if the negative
         * value was to be stored because the next time it is used the value is
         * squared.
         */
    	tid += blockDim.x * gridDim.x + blockDim.y * gridDim.y;
    }
}




//Input: dev_r_vec, dev_g_vec, dev_b_vec: vectors containing the R,G,B components of the input image
//		 gray_size: amount of pixels in the RGB vector / 3
//Output: dev_gray_image: a vector containing the gray-scale pixels of the resulting image

// CUDA kernel to convert an image to gray-scale
//gray-image's memory needs to be pre-allocated
__global__ void rgb_img_to_gray( byte * dev_r_vec, byte * dev_g_vec, byte * dev_b_vec, byte * dev_gray_image, int gray_size)
{
    //Get the id of thread within a block
	int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
	int tid_y = threadIdx.y + blockIdx.y * blockDim.y;

	//simple linearization of 2D space
	int tid = abs(tid_x - tid_y);

	//pixel-wise operation on the R,G,B vectors
	while(tid < gray_size)
	{
		//r, g, b pixels
		byte p_r = dev_r_vec[tid];
		byte p_g = dev_g_vec[tid];
		byte p_b = dev_b_vec[tid];

		//Formula accordidev_ng to: https://stackoverflow.com/questions/17615963/standard-rgb-to-grayscale-conversion
		dev_gray_image[tid] = 0.30 * p_r + 0.59*p_g + 0.11*p_b;
    	tid += blockDim.x * gridDim.x + blockDim.y * gridDim.y;

	}
}



int main ( int argc, char** argv )
{
		//actual computation
		struct timeval comp_start_load_img, comp_end_load_img;

		gettimeofday(&comp_start_load_img, NULL);
		if(argc < 2)
		{
			printf("You did not provide any input image name. Please, provide an input image name and retry. \n");
			return -2;
		}

		bool intermediate_output = false;

		//###########1. STEP - LOAD THE IMAGE, ITS HEIGHT, WIDTH AND CONVERT IT TO RGB FORMAT#########

		//Specify the input image. Formats supported: png, jpg, GIF.
		//const char * fileInputName = "imgs_in/hua_hua.jpg";
		//Example argv[1] = "imgs_in/hua_hua.pjg";
		const char * fileInputName = argv[1];
		const char * spaceDiv = " ";
		const char * fileOutputRGB = "imgs_out/image.rgb";
		const char *pngStrings[4] = {"convert ", fileInputName, spaceDiv, fileOutputRGB};
		const char * strPngToRGB = arrayStringsToString(pngStrings, 4, STRING_BUFFER_SIZE);

		printf("Loading input image [%s] \n", fileInputName);

		gettimeofday(&comp_end_load_img, NULL);

		struct timeval i_o_start_load_img, i_o_end_load_img;
		gettimeofday(&i_o_start_load_img, NULL);
		//execute the conversion from PNG to RGB, as that format is required for the program
		int status_conversion = system(strPngToRGB);
		gettimeofday(&i_o_end_load_img, NULL);

		struct timeval comp_start_img_conv, comp_end_img_conv;

		gettimeofday(&comp_start_img_conv, NULL);
		if(status_conversion != 0)
		{
			printf("ERROR! Conversion of input PNG image to RGB was not successful. Program aborting.\n");
			return -1;
		}
		printf("Converted input image to RGB [%s] \n", fileOutputRGB);

		//get the height and width of the input image
		int width = 0;
		int height = 0;

		getImageSize(fileInputName, &width, &height);

		printf("Size of the loaded image: width=%d height=%d \n", width, height);

		//Three dimensions because the input image is in colored format(R,G,B)
		int rgb_size = width * height * 3;
		printf("Total amount of pixels in RGB input image is [%d] \n", rgb_size);
		//Used as a buffer for all pixels of the image
		byte * rgb_image;

		//Load up the input image in RGB format into one single flattened array (rgbImage)
		readFile(fileOutputRGB, &rgb_image, rgb_size);

		//########2. step - convert RGB image to gray-scale
	    int gray_size = rgb_size / 3;
	    byte * rVector, * gVector, * bVector;

	    //now take the RGB image vector and create three separate arrays for the R,G,B dimensions
	    getDimensionFromRGBVec(0, rgb_image,  &rVector, gray_size);
	    getDimensionFromRGBVec(1, rgb_image,  &gVector, gray_size);
	    getDimensionFromRGBVec(2, rgb_image,  &bVector, gray_size);

	    //allocate memory on the device for the r,g,b vectors
	    byte * dev_r_vec, * dev_g_vec, * dev_b_vec;
	    byte * dev_gray_image;

		gettimeofday(&comp_end_img_conv, NULL);
		struct timeval start_alloc_rgb, end_alloc_rgb;
		gettimeofday(&start_alloc_rgb, NULL);

	    HANDLE_ERROR ( hipMalloc((void **)&dev_r_vec , gray_size*sizeof(byte)));
	    HANDLE_ERROR ( hipMalloc((void **)&dev_g_vec, gray_size*sizeof(byte)));
	    HANDLE_ERROR ( hipMalloc((void **)&dev_b_vec, gray_size*sizeof(byte)));

	    //copy the content of the r,g,b vectors from the host to the device
	    HANDLE_ERROR (hipMemcpy (dev_r_vec , rVector , gray_size*sizeof(byte), hipMemcpyHostToDevice));
	    HANDLE_ERROR (hipMemcpy (dev_g_vec , gVector , gray_size*sizeof(byte), hipMemcpyHostToDevice));
	    HANDLE_ERROR (hipMemcpy (dev_b_vec , bVector , gray_size*sizeof(byte), hipMemcpyHostToDevice));
	    //allocate memory on the device for the output gray image
	    HANDLE_ERROR ( hipMalloc((void **)&dev_gray_image, gray_size*sizeof(byte)));

		gettimeofday(&end_alloc_rgb, NULL);

		struct timeval comp_start_rgb_to_gray, comp_end_rgb_to_gray;
		gettimeofday(&comp_start_rgb_to_gray, NULL);

	    //actually run the kernel to convert input RGB file to gray-scale
	    rgb_img_to_gray <<< width, height>>> (dev_r_vec, dev_g_vec, dev_b_vec, dev_gray_image, gray_size) ;

	    //TODO: use malloc instead of [gray_size]
	    byte gray_image[gray_size];

		gettimeofday(&comp_end_rgb_to_gray, NULL);

		struct timeval start_gray_vec_copy, end_gray_vec_copy;

		gettimeofday(&start_gray_vec_copy, NULL);
	    //Now take the device gray vector and bring it back to the host
	    HANDLE_ERROR (hipMemcpy(gray_image , dev_gray_image , gray_size*sizeof(byte) , hipMemcpyDeviceToHost));
		gettimeofday(&end_gray_vec_copy, NULL);

		struct timeval comp_start_str_alloc, comp_end_str_alloc;

		gettimeofday(&comp_start_str_alloc, NULL);
		char str_width[100];
		sprintf(str_width, "%d", width);

		char str_height[100];
		sprintf(str_height, "%d", height);

		gettimeofday(&comp_end_str_alloc, NULL);

		if(intermediate_output)
		{
			 //let's see what's in there, shall we?
			const char * file_gray = "imgs_out/img_gray.gray";
			writeFile(file_gray, gray_image, gray_size);
			printf("Total amount of pixels in gray-scale image is [%d] \n", gray_size);
			const char * file_png_gray = "imgs_out/img_gray.png";

			const char * pngConvertGray[8] = {"convert -size ", str_width, "x", str_height, " -depth 8 ", file_gray, spaceDiv, file_png_gray};
			char * strGrayToPNG = arrayStringsToString(pngConvertGray, 8, STRING_BUFFER_SIZE);
			system(strGrayToPNG);
			printf("Converted gray image to PNG [%s]\n", file_png_gray);
		}


		struct timeval start_free_rgb, end_free_rgb;

		gettimeofday(&start_free_rgb, NULL);
	    hipFree (dev_r_vec);
	    hipFree (dev_g_vec);
		hipFree (dev_b_vec);
		gettimeofday(&end_free_rgb, NULL);

		//######################3. Step - Compute vertical and horizontal gradient ##########

		//###Compute the HORIZONTAL GRADIENT#####

		struct timeval comp_start_alloc_h_vec, comp_end_alloc_h_vec;

		gettimeofday(&comp_start_alloc_h_vec, NULL);
   	    //host horizontal kernel
		int sobel_h[] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
		int * dev_sobel_h;
   	    byte * dev_sobel_h_res;
		gettimeofday(&comp_end_alloc_h_vec, NULL);

		struct timeval start_h_vec_alloc, end_h_vec_alloc;

		gettimeofday(&start_h_vec_alloc, NULL);

		//allocate memory for device horizontal kernel
		HANDLE_ERROR ( hipMalloc((void **)&dev_sobel_h , SOBEL_OP_SIZE*sizeof(int)));

		//copy the content of the host horizontal kernel to the device horizontal kernel
	    HANDLE_ERROR (hipMemcpy (dev_sobel_h , sobel_h , SOBEL_OP_SIZE*sizeof(int) , hipMemcpyHostToDevice));

	    //allocate memory for the resulting horizontal gradient on the device
		HANDLE_ERROR ( hipMalloc((void **)&dev_sobel_h_res , gray_size*sizeof(byte)));

		gettimeofday(&end_h_vec_alloc, NULL);

		struct timeval comp_start_horiz_grad, comp_end_horiz_grad;

		gettimeofday(&comp_start_horiz_grad, NULL);
		//perform horizontal gradient calculation for every pixel
		it_conv <<< width, height>>> (dev_gray_image, gray_size, width, dev_sobel_h, dev_sobel_h_res);

		//fixed segmentation fault when processing large images
		byte* sobel_h_res = (byte*) malloc(gray_size * sizeof(byte));

		gettimeofday(&comp_end_horiz_grad, NULL);

		//copy the resulting horizontal array from device to host

		struct timeval start_h_vec_copy, end_h_vec_copy;

		gettimeofday(&start_h_vec_copy, NULL);
	    HANDLE_ERROR (hipMemcpy(sobel_h_res , dev_sobel_h_res , gray_size*sizeof(byte) , hipMemcpyDeviceToHost));
	    gettimeofday(&end_h_vec_copy, NULL);

		struct timeval start_h_vec_free, end_h_vec_free;

		gettimeofday(&start_h_vec_free, NULL);
	    //free-up the memory for the vectors allocated
	    hipFree(dev_sobel_h);
	    gettimeofday(&end_h_vec_free, NULL);

	    const char * strGradToPNG;

	    if(intermediate_output)
	    {
			//output the horizontal axis-gradient to a file
			const char * file_out_h_grad = "imgs_out/sobel_horiz_grad.gray";
			//Output the horizontal axis' gradient calculation
			writeFile(file_out_h_grad, sobel_h_res, gray_size);
			printf("Output horizontal gradient to [%s] \n", file_out_h_grad);
			const char * fileHorGradPNG = "imgs_out/sobel_horiz_grad.png";
			printf("Converted horizontal gradient: ");
			printf("[%s] \n", fileHorGradPNG);
			//Convert the output file to PNG
			const char * pngConvertHor[8] = {"convert -size ", str_width, "x", str_height, " -depth 8 ", file_out_h_grad, spaceDiv, fileHorGradPNG};
			const char * strGradToPNG = arrayStringsToString(pngConvertHor, 8, STRING_BUFFER_SIZE);
			system(strGradToPNG);
	    }

		struct timeval comp_start_alloc_v_grad, comp_end_alloc_v_grad;
		gettimeofday(&comp_start_alloc_v_grad, NULL);
		//####Compute the VERTICAL GRADIENT#####
	    int sobel_v[] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
		int * dev_sobel_v;
		byte * dev_sobel_v_res;
		gettimeofday(&comp_end_alloc_v_grad, NULL);

		struct timeval start_v_vec_alloc, end_v_vec_alloc;

		gettimeofday(&start_v_vec_alloc, NULL);

		//allocate memory for device vertical kernel
		HANDLE_ERROR (hipMalloc((void **)&dev_sobel_v , SOBEL_OP_SIZE*sizeof(int)));

		//copy the content of the host vertical kernel to the device vertical kernel
		HANDLE_ERROR (hipMemcpy (dev_sobel_v , sobel_v , SOBEL_OP_SIZE*sizeof(int) , hipMemcpyHostToDevice));

		//allocate memory for the resulting vertical gradient on the device
		HANDLE_ERROR (hipMalloc((void **)&dev_sobel_v_res , gray_size*sizeof(byte)));

		gettimeofday(&end_v_vec_alloc, NULL);

		struct timeval comp_start_vert_grad, comp_end_vert_grad;
		gettimeofday(&comp_start_vert_grad, NULL);

		//perform vertical gradient calculation for every pixel
		it_conv <<<width, height>>> (dev_gray_image, gray_size, width, dev_sobel_v, dev_sobel_v_res);

		//copy the resulting vertical array from device back to host
		//fixed segmentation fault issue with big images
		byte* sobel_v_res = (byte*) malloc(gray_size * sizeof(byte));

		gettimeofday(&comp_end_vert_grad, NULL);

		struct timeval start_v_vec_copy, end_v_vec_copy;

		gettimeofday(&start_v_vec_copy, NULL);
		HANDLE_ERROR (hipMemcpy(sobel_v_res , dev_sobel_v_res , gray_size*sizeof(byte) , hipMemcpyDeviceToHost));
		gettimeofday(&end_v_vec_copy, NULL);

		//free-up the memory for the vectors allocated
		struct timeval start_v_vec_free, end_v_vec_free;
		gettimeofday(&start_v_vec_free, NULL);
		hipFree(dev_sobel_v);
		gettimeofday(&end_v_vec_free, NULL);

		struct timeval comp_start_countour_alloc, comp_end_countour_alloc;
		gettimeofday(&comp_start_countour_alloc, NULL);

		if(intermediate_output)
		{
			const char * file_out_v_grad = "imgs_out/sobel_vert_grad.gray";

			//Output the vertical axis' gradient calculated
			writeFile(file_out_v_grad, sobel_v_res, gray_size);

			printf("Output vertical gradient to [%s] \n", file_out_v_grad);
			const char * fileVerGradPNG = "imgs_out/sobel_vert_grad.png";

			const char * pngConvertVer[8] = {"convert -size ", str_width, "x", str_height, " -depth 8 ", file_out_v_grad, spaceDiv, fileVerGradPNG};

			strGradToPNG = arrayStringsToString(pngConvertVer, 8, STRING_BUFFER_SIZE);
			system(strGradToPNG);
		}


		//#############4. Step - Compute the countour by putting together the vertical and horizontal gradients####

		//allocate device memory for the final vector containing the countour

		byte * dev_countour_img;

		gettimeofday(&comp_end_countour_alloc, NULL);

		struct timeval start_countour_alloc, end_countour_alloc;

		gettimeofday(&start_countour_alloc, NULL);
		HANDLE_ERROR ( hipMalloc((void **)&dev_countour_img , gray_size*sizeof(byte)));
		gettimeofday(&end_countour_alloc, NULL);

		struct timeval comp_start_countour_merge, comp_end_countour_merge;

		gettimeofday(&comp_start_countour_merge, NULL);
		contour <<< width, height>>> (dev_sobel_h_res, dev_sobel_v_res, gray_size, dev_countour_img);
		//copy the resulting countour image from device back to host
		byte countour_img[gray_size];
		gettimeofday(&comp_end_countour_merge, NULL);

		struct timeval start_countour_copy, end_countour_copy;

		gettimeofday(&start_countour_copy, NULL);
		HANDLE_ERROR (hipMemcpy(countour_img, dev_countour_img, gray_size*sizeof(byte) , hipMemcpyDeviceToHost));
		gettimeofday(&end_countour_copy, NULL);

		struct timeval start_free_countour, end_free_countour;

		gettimeofday(&start_free_countour, NULL);
		//free-up all the memory from the allocate vectors
	    hipFree(dev_sobel_h_res);
	    hipFree(dev_sobel_v_res);
	    hipFree(dev_countour_img);
	    gettimeofday(&end_free_countour, NULL);

	    //######Display the resulting countour image

		struct timeval comp_start_countour_str_alloc, comp_end_countour_str_alloc;
		gettimeofday(&comp_start_countour_str_alloc, NULL);
		const char * file_sobel_out = "imgs_out/sobel_countour.gray";
		const char * file_sobel_png = "imgs_out/sobel_countour.png";
		const char * pngConvertContour[8] = {"convert -size ", str_width, "x", str_height, " -depth 8 ", file_sobel_out, spaceDiv, file_sobel_png};
		const char * strSobelToPNG = arrayStringsToString(pngConvertContour, 8, STRING_BUFFER_SIZE);
		gettimeofday(&comp_end_countour_str_alloc, NULL);


		struct timeval i_o_start_write_gray_countour, i_o_end_write_gray_countour;

		gettimeofday(&i_o_start_write_gray_countour, NULL);
		writeFile(file_sobel_out, countour_img, gray_size);
		gettimeofday(&i_o_end_write_gray_countour, NULL);

		printf("Output countour to [%s] \n", file_sobel_out);

		//actually execute the conversion from PNG to RGB, as that format is required for the program
		struct timeval i_o_start_write_img, i_o_end_write_img;
		gettimeofday(&i_o_start_write_img, NULL);
		system(strSobelToPNG);
		gettimeofday(&i_o_end_write_img, NULL);

		printf("Converted countour: [%s] \n", file_sobel_png);
		printf("SUCCESS! Successfully applied Sobel filter to the input image!\n");
		//printf("file loading and writing time: [%d] ms", write_load_total_time);


		//#############5. Step - Display the elapsed time in the different parts of the code

		//##GPU memory movements (hipMalloc, cudaMemCpy, hipFree) ##
		double time_alloc_rgb = compute_elapsed_time(start_alloc_rgb, end_alloc_rgb);
		double time_free_rgb = compute_elapsed_time(start_free_rgb, end_free_rgb);
		double time_copy_gray = compute_elapsed_time(start_gray_vec_copy, end_gray_vec_copy);
		//horizontal vector operations
		double time_alloc_h_vec = compute_elapsed_time(start_h_vec_alloc, end_h_vec_alloc);
		double time_copy_h_vec = compute_elapsed_time(start_h_vec_copy, end_h_vec_copy);
		double time_free_h_vec = compute_elapsed_time(start_h_vec_free, end_h_vec_free);
		//vertical vector operations
		double time_alloc_v_vec = compute_elapsed_time(start_v_vec_alloc, end_v_vec_alloc);
		double time_copy_v_vec = compute_elapsed_time(start_v_vec_copy, end_v_vec_copy);
		double time_free_v_vec = compute_elapsed_time(start_v_vec_free, end_v_vec_free);
		//countour image operations
		double time_alloc_countour = compute_elapsed_time(start_countour_alloc, end_countour_alloc);
		double time_copy_countour = compute_elapsed_time(start_countour_copy, end_countour_copy);
		double time_free_countour = compute_elapsed_time(start_free_countour, end_free_countour);

		double total_time_gpu_mem = time_alloc_rgb + time_free_rgb + time_copy_gray + time_alloc_h_vec + time_copy_h_vec + time_free_h_vec +
							  time_alloc_v_vec + time_copy_v_vec + time_free_v_vec + time_alloc_countour + time_copy_countour + time_free_countour;

		printf("Time spent on GPU memory operations: [%f] ms\n", total_time_gpu_mem);

		//##Actual GPU computation##
		double comp_time_load_img = compute_elapsed_time(comp_start_load_img, comp_end_load_img);
		double comp_time_convert_img = compute_elapsed_time(comp_start_img_conv, comp_end_img_conv);
		double comp_time_rgb_to_gray = compute_elapsed_time(comp_start_rgb_to_gray, comp_end_rgb_to_gray);
		double comp_time_str_alloc = compute_elapsed_time(comp_start_str_alloc, comp_end_str_alloc);
		double comp_time_h_alloc = compute_elapsed_time(comp_start_alloc_h_vec, comp_end_alloc_h_vec);
		double comp_time_h_grad = compute_elapsed_time(comp_start_horiz_grad, comp_end_horiz_grad);
		double comp_time_v_alloc = compute_elapsed_time(comp_start_alloc_v_grad, comp_end_alloc_v_grad);
		double comp_time_v_grad = compute_elapsed_time(comp_start_vert_grad, comp_end_vert_grad);
		double comp_time_count_alloc = compute_elapsed_time(comp_start_countour_alloc, comp_end_countour_alloc);
		double comp_time_count_merge = compute_elapsed_time(comp_start_countour_merge, comp_end_countour_merge);
		double comp_time_count_str_alloc = compute_elapsed_time(comp_start_countour_str_alloc, comp_end_countour_str_alloc);

		double total_time_gpu_comp = comp_time_load_img + comp_time_convert_img + comp_time_rgb_to_gray + comp_time_str_alloc + comp_time_h_alloc +
						comp_time_h_grad + comp_time_v_alloc + comp_time_v_grad + comp_time_count_alloc + comp_time_count_merge + comp_time_count_str_alloc;

		printf("Time spent on GPU computation: [%f] ms\n", total_time_gpu_comp);

		//##Input/Output over the disk (image loading and final image writing)##
		double i_o_time_load_img = compute_elapsed_time(i_o_start_load_img, i_o_end_load_img);
		double i_o_time_write_gray_countour = compute_elapsed_time(i_o_start_write_gray_countour, i_o_end_write_gray_countour);
		double i_o_time_write_img = compute_elapsed_time(i_o_start_write_img, i_o_end_write_img);

		double total_time_i_o = i_o_time_load_img + i_o_time_write_gray_countour + i_o_time_write_img;

		printf("Time spent on I/O operations from/to disk: [%f] ms\n", total_time_i_o);

		//##Overall time spent in the program
		double overall_total_time = total_time_gpu_comp + total_time_gpu_mem + total_time_i_o;

		printf("Overall time spent in program [%f] ms \n", overall_total_time);

	    return 0;

}

//Don't forget to clean up the device memory!!
