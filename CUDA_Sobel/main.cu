#include "hip/hip_runtime.h"
#include <stdio.h>

#include "file_operations.c"

#define STRING_BUFFER_SIZE 1024




#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
      {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}


//Input: - rgb image contained in the 'rgb' array
//		 - buffer size: the size of the RGB image
//Output: gray, an array containing the gray-scale image
int rgbToGray(byte *rgbImage, byte **grayImage, int gray_size)
{
    // Take size for gray image and allocate memory. Just one dimension for gray-scale image
    *grayImage = (byte*) malloc(sizeof(byte) * gray_size);

    // Make pointers for iteration
    byte *p_rgb = rgbImage;
    byte *p_gray = *grayImage;

    // Calculate the value for every pixel in gray
    for(int i=0; i<gray_size; i++)
    {
    	//Formula according to: https://stackoverflow.com/questions/17615963/standard-rgb-to-grayscale-conversion
        *p_gray = 0.30*p_rgb[0] + 0.59*p_rgb[1] + 0.11*p_rgb[2];
        p_rgb += 3;
        p_gray++;
    }

    return gray_size;
}




// CUDA //kernel to convert an image to gray-scale
//gray-image's memory needs to be pre-allocated
__global__ void rgb_img_to_gray( byte * dev_r_vec, byte * dev_g_vec, byte * dev_b_vec, byte * dev_gray_image, int gray_size)
{
    //Get the id of thread within a block
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	while(tid < gray_size)
	{
		//r, g, b pixels
		byte p_r = dev_r_vec[tid];
		byte p_g = dev_g_vec[tid];
		byte p_b = dev_b_vec[tid];

		//Formula according to: https://stackoverflow.com/questions/17615963/standard-rgb-to-grayscale-conversion
		dev_gray_image[tid] = 0.30 * p_r + 0.59*p_g + 0.11*p_b;
    	tid += blockDim.x * gridDim.x;
	}
}








int main (void)
{
	//###########1. STEP - LOAD THE IMAGE, ITS HEIGHT, WIDTH AND CONVERT IT TO RGB FORMAT#########

		//Specify the input image. Formats supported: png, jpg, GIF.
		const char * fileInputName = "imgs_in/lena.png";

		const char * spaceDiv = " ";
		const char * fileOutputRGB = "imgs_out/image.rgb";
		const char *pngStrings[4] = {"convert ", fileInputName, spaceDiv, fileOutputRGB};
		const char * strPngToRGB = arrayStringsToString(pngStrings, 4, STRING_BUFFER_SIZE);

		printf("Loading input image [%s] \n", fileInputName);

		//actually execute the conversion from PNG to RGB, as that format is required for the program
		int status_conversion = system(strPngToRGB);

		if(status_conversion != 0)
		{
			printf("Conversion of input PNG image to RGB was not successful. Program aborting.");
			return -1;
		}
		printf("Converted input image to RGB [%s] \n", fileOutputRGB);

		//get the height and width of the input image
		int width = 0;
		int height = 0;

		getImageSize(fileInputName, &width, &height);

		printf("Size of the loaded image : width=%d height=%d \n", width, height);

		//Three dimensions because the input image is in colored format(R,G,B)
		int rgb_size = width * height * 3;
		printf("Total amount of pixels in RGB input image is [%d] \n", rgb_size);
		//Used as a buffer for all pixels of the image
		byte * rgb_image;

		//Load up the input image in RGB format into one single flattened array (rgbImage)
		readFile(fileOutputRGB, &rgb_image, rgb_size);

		//########2. step - convert RGB image to gray-scale

	    int gray_size = rgb_size / 3;

	    byte * rVector, * gVector, * bVector;

	    //now take the RGB image vector and create three separate arrays for the R,G,B dimensions
	    getDimensionFromRGBVec(0, rgb_image,  &rVector, gray_size);
	    getDimensionFromRGBVec(1, rgb_image,  &gVector, gray_size);
	    getDimensionFromRGBVec(2, rgb_image,  &bVector, gray_size);

	    //allocate memory on the device for the r,g,b vectors
	    byte * dev_r_vec, * dev_g_vec, * dev_b_vec;
	    HANDLE_ERROR ( hipMalloc((void **)&dev_r_vec , gray_size*sizeof(byte)));
	    HANDLE_ERROR ( hipMalloc((void **)&dev_g_vec, gray_size*sizeof(byte)));
	    HANDLE_ERROR ( hipMalloc((void **)&dev_b_vec, gray_size*sizeof(byte)));

	    //copy the content of the r,g,b vectors from the host to the device
	    HANDLE_ERROR (hipMemcpy (dev_r_vec , rVector , gray_size*sizeof(byte), hipMemcpyHostToDevice));
	    HANDLE_ERROR (hipMemcpy (dev_g_vec , gVector , gray_size*sizeof(byte), hipMemcpyHostToDevice));
	    HANDLE_ERROR (hipMemcpy (dev_b_vec , bVector , gray_size*sizeof(byte), hipMemcpyHostToDevice));

	    //allocate memory on the device for the output gray image
	    byte * dev_gray_image;
	    HANDLE_ERROR ( hipMalloc((void **)&dev_gray_image, gray_size*sizeof(byte)));

	    //actually run the kernel to convert input RGB file to gray-scale
	    rgb_img_to_gray <<< width, height>>> (dev_r_vec, dev_g_vec, dev_b_vec, dev_gray_image, gray_size) ;

	    byte gray_image[gray_size];

	    //Now take the device gray vector and bring it back to the host
	    HANDLE_ERROR (hipMemcpy(gray_image , dev_gray_image , gray_size*sizeof(byte) , hipMemcpyDeviceToHost));

	    //let's see what's in there, shall we?
	    const char * file_gray = "imgs_out/img_gray.gray";

		writeFile(file_gray, gray_image, gray_size);
		printf("Total amount of pixels in gray-scale image is [%d] \n", gray_size);

		const char * file_png_gray = "imgs_out/img_gray.png";

		char str_width[100];
		sprintf(str_width, "%d", width);

		char str_height[100];
		sprintf(str_height, "%d", height);

		const char * pngConvertGray[8] = {"convert -size ", str_width, "x", str_height, " -depth 8 ", file_gray, spaceDiv, file_png_gray};
		char * strGrayToPNG = arrayStringsToString(pngConvertGray, 8, STRING_BUFFER_SIZE);
		system(strGrayToPNG);

		printf("Converted gray image to PNG [%s]\n", file_png_gray);

		//######################3. Step - Compute vertical and horizontal gradient ##########



}

//Don't forget to clean up the device memory!!
